//#include "kernel.cuh"
//#include "cuda_runtime.h"

#include <hip/hip_runtime.h>
#include <iostream>
//#include "device_launch_parameters.h"
//#include "device_functions.h"
//#include <stdio.h>
//#include "constsmacros.h"
#include <stdlib.h>
//#include <math.h>
//#include "ImageLoading.cu"
////#include "CUDAArray.cuh"
//#include <float.h>
//#include "OrientationField.cu"
//#include "Convolution.cu"
//
//#define M_PI 3.14159265358979323846
//#define Pi4 (M_PI / 4)
//
//__device__ __host__
//Point NewPoint(int x, int y)
//{
//	Point newP;
//	newP.x = x;
//	newP.y = y;
//	return newP;
//}
//
//__device__ void AddMinutiae(CUDAArray<Minutiae>* minutiaes, Minutiae minutiae, int* indexOfMinutiae)
//{
//	minutiaes->SetAt(0, *indexOfMinutiae, minutiae);
//	*indexOfMinutiae += 1;
//}
//
//__device__ bool OutOfImage(CUDAArray<float> image, int x, int y, int partX, int partY)
//{
//	return (x < 0) || (y < 0) || (y >= image.Width) || (x >= image.Height);
//	//return (x < blockIdx.x * partX) || (y < blockIdx.y * partY) || (x >= (blockIdx.x + 1) * partX) || (y >= (blockIdx.y + 1) * partY) || (y >= image.Width) || (x >= image.Height);
//}
//
//__device__ void NewSection(int x, int y, Direction direction, CUDAArray<float> image, CUDAArray<float> orientationField, 
//	Point* section, float* sectionAngle, int* centerSection, bool* flag, int size, int partX, int partY)
//{
//	int lengthWings = size / 2;
//
//	for (int i = 0; i < size; i++)
//	{
//		section[i] = NewPoint(-1, -1);
//
//	}
//
//	int lEnd = lengthWings;
//	int rEnd = lEnd;
//
//	bool rightE = false;
//	bool leftE = false;
//
//	float angle = -orientationField.At(y, x);
//	angle += M_PI_2;
//
//	section[lengthWings] = NewPoint(x, y);
//
//	for (int i = 1; i <= lengthWings; i++)
//	{
//		int xs = (int)(x - i * cos(angle));
//		int ys = (int)(y + i * sin(angle) + 0.95);
//		int xe = (int)(x + i * cos(angle) + 0.95);
//		int ye = (int)(y - i * sin(angle));
//
//		if (!OutOfImage(image, xs, ys, partX, partY) && (image.At(ys, xs) < 20) && !rightE)
//		{
//			section[lengthWings - i] = NewPoint(xs, ys);
//			rEnd--;
//		}
//		else
//		{
//			rightE = true;
//		}
//
//		if (!OutOfImage(image, xe, ye, partX, partY) && (image.At(ye, xe) < 20) && !leftE)
//		{
//			section[lengthWings + i] = NewPoint(xe, ye);
//			lEnd++;
//		}
//		else
//		{
//			leftE = true;
//		}
//
//		*centerSection = (lEnd + rEnd) / 2;
//	}
//
//	x = section[*centerSection].x;
//	y = section[*centerSection].y;
//
//	angle = -orientationField.At(y, x);
//	angle += (float) direction * M_PI;
//	if (angle < 0) angle += 2.0 * M_PI;
//
//	if (abs(*sectionAngle - angle) > 0.2 && abs(*sectionAngle - angle) < 6) angle += M_PI;
//	while (angle > 2.0 * M_PI) angle -= 2 * M_PI;
//
//	*sectionAngle = angle;
//}
//
//
//
//__device__ bool CheckAndDeleteFalseMinutia(Minutiae minutia)
//{
//	return false;
//}
//
//__device__ void MakeStep(int* x, int* y, CUDAArray<float> image, Point* section, int* centerSection, float* sectionAngle, int step, int partX, int partY)
//{
//	*x = section[*centerSection].x;
//	*y = section[*centerSection].y;
//
//	float dx = (float)*x + (float)step * cos(*sectionAngle);
//	float dy = (float)*y - (float)step * sin(*sectionAngle);
//
//	*x = (int)(dx >= 0 ? dx + 0.5 : dx - 0.5);
//	*y = (int)(dy >= 0 ? dy + 0.5 : dy - 0.5);
//
//	if (OutOfImage(image, *x, *y, partX, partY))
//	{
//		*x = -1;
//		*y = -1;
//	}
//}
//
//__device__ MinutiaeType CheckStopCriteria(int x, int y, CUDAArray<float> image, CUDAArray<bool> visited, int threshold = 20)
//{
//	if (visited.At(y, x))
//		return Intersection;
//	if (image.At(y, x) > threshold)
//		return LineEnding;
//
//	return NotMinutiae;
//}
//
//__device__ void Paint(CUDAArray<float> image, CUDAArray<bool> visited, Point* oldSection, Point* section, int size, int partX, int partY)
//{
//	Point queue[40];
//
//	int shift = 0; // 30 * (blockIdx.x * gridDim.x + blockIdx.y);
//	int rPointer = 0, lPointer = 0;
//	//queue = (Point*)malloc(32 * 32 * sizeof(Point));
//	Point v1, v2;
//
//	int x1 = -1, x2 = -1, y1 = -1, y2 = -1, x_a, y_a;
//
//	for (int i = 0; i < size; i++)
//	{
//		if (oldSection[i].x == -1) continue;
//
//		if (x1 == -1)
//		{
//			x1 = oldSection[i].x;
//			y1 = oldSection[i].y;
//		}
//
//		x2 = oldSection[i].x;
//		y2 = oldSection[i].y;
//
//		visited.SetAt(y2, x2, true);
//		queue[shift + rPointer] = oldSection[i];
//		rPointer++;
//	}
//
//	v1 = NewPoint(x2 - x1, y2 - y1);
//	x_a = x1;
//	y_a = y1;
//
//	x1 = -1;
//	y1 = -1;
//	x2 = -1;
//	y2 = -1;
//
//	for (int i = 0; i < size; i++)
//	{
//		if (section[i].x == -1) continue;
//
//		if (x1 == -1)
//		{
//			x1 = section[i].x;
//			y1 = section[i].y;
//		}
//
//		x2 = section[i].x;
//		y2 = section[i].y;
//
//		visited.SetAt(y2, x2, true);
//	}
//
//	v2 = NewPoint(x2 - x1, y2 - y1);
//
//	if (v1.x*v2.x + v1.y*v2.y < 0)
//	{
//		x1 = x2;
//		y1 = y2;
//		v1.x = -v1.x;
//		v1.y = -v1.y;
//	}
//
//	while (abs(rPointer - lPointer) > 0)
//	{
//		for (int i = -2; i < 3; i++)
//			for (int j = -2; j < 3; j++)
//			{
//				if (i == 0 && j == 0) continue;
//
//				int x = queue[lPointer].x + i;
//				int y = queue[lPointer].y + j;
//
//				if (OutOfImage(image, x, y, partX, partY) || visited.At(y, x) || image.At(y, x) > 20) continue;
//
//				Point pointV1 = NewPoint(x_a - x, y_a - y);
//				Point pointV2 = NewPoint(x1 - x, y1 - y);
//
//				int skew1 = v1.x*pointV1.y - pointV1.x*v1.y >= 0 ? 1 : -1;
//				int skew2 = v2.x*pointV2.y - pointV2.x*v2.y >= 0 ? 1 : -1;
//
//				if (skew1*skew2 < 0)
//				{
//					queue[shift + rPointer] = NewPoint(x, y);
//					rPointer++;
//					visited.SetAt(y, x, true);
//					if (rPointer == 40) rPointer = 0;
//				}
//			}
//
//		lPointer++;
//		if (lPointer == 40) lPointer = 0;
//	}
//}
//
//__device__ void FollowLine(int x, int y, Direction direction, CUDAArray<float> image, CUDAArray<float> orientationField,
//	CUDAArray<bool> visited, CUDAArray<Minutiae> minutiaes,	Point* section, float* sectionAngle, 
//	int* centerSection, bool* flag, int size, int step, int partX, int partY, int* indexOfMinutiae)
//{
//	NewSection(x, y, direction, image, orientationField, section, sectionAngle, 
//		centerSection, flag, size, partX, partY);
//	if (section[*centerSection].x == -1) return;
//
//	MinutiaeType type;
//
//	Point* oldSection = new Point[size];
//
//	do
//	{		
//		for (int i = 0; i < size; i++){
//			oldSection[i] = section[i];
//		}
//
//		MakeStep(&x, &y, image, section, centerSection, sectionAngle, step, partX, partY);
//		
//		if (x == -1) return;
//
//		NewSection(x, y, direction, image, orientationField, section, sectionAngle, centerSection, flag, size, partX, partY);
//		if (section[*centerSection].x == -1) return;
//
//		type = CheckStopCriteria(x, y, image, visited);
//
//		Paint(image, visited, oldSection, section, size, partX, partY);
//	} while (type == NotMinutiae);
//
//	Minutiae possMinutiae;
//	possMinutiae.x = x;
//	possMinutiae.y = y;
//	possMinutiae.angle = *sectionAngle;
//	possMinutiae.type = type;
//
//	//printf("Minutia. x = %d y = %d type = %d\n", possMinutiae.x, possMinutiae.y, possMinutiae.type);
//
//	//if (IsDuplicate(possMinutiae)) return;
//
//	if (!CheckAndDeleteFalseMinutia(possMinutiae))
//	{
//		//printf("Minutia. x = %d y = %d type = %d\n", possMinutiae.x, possMinutiae.y, possMinutiae.type);
//		AddMinutiae(&minutiaes, possMinutiae, indexOfMinutiae);
//		//printf("%d %d: minutiae detected. x = %d; y = %d; type = %d\n", blockIdx.x, blockIdx.y, possMinutiae.x, possMinutiae.y, possMinutiae.type);
//	}
//}
//
//__global__ void FindMinutia(CUDAArray<float> image, CUDAArray<float> orientationField, CUDAArray<bool> visited,
//	CUDAArray<Minutiae> minutiaes, const int size, const int step, int colorThreshold = 15)
//{
//	Point* section = new Point[size];
//	float sectionAngle;
//	int centerSection;
//	bool flag  = false;
//
//	int partX = 32; //image.Height / gridDim.x;
//	int partY = 32; //image.Width / gridDim.y;
//
//	int indexOfMinutiae = 0; // blockIdx.x * image.Height + blockIdx.y * defaultThreadCount;
//
//	//printf("%d %d %d\n", blockIdx.x, blockIdx.y, threadIdx.x);
//
//	//if (blockIdx.x == 7 && blockIdx.y == 7)
//	//for (int i = blockIdx.x * partX; i < (blockIdx.x + 1) * partX; i++)
//	//	for (int j = blockIdx.y * partY; j < (blockIdx.y + 1) * partY; j++)
//	for (int i = 0; i < image.Width; i++)
//		for (int j = 0; j < image.Height; j++)
//		{
//			if (OutOfImage(image, i, j, partX, partY))
//			{
//				//printf("Tu-tu. %d %d\n", i, j);
//				continue;
//			}
//
//			if ((image.At(j, i) >= colorThreshold) || visited.At(j, i)) continue;
//			
//			sectionAngle = -orientationField.At(j, i);
//			if (sectionAngle < 0) sectionAngle += 2.0 * M_PI;
//			FollowLine(i, j, Forward, image, orientationField, visited, minutiaes, 
//				section, &sectionAngle, &centerSection, &flag, size, step, partX, partY, &indexOfMinutiae);
//
//			flag = false;
//
//			sectionAngle = -orientationField.At(j, i) + M_PI;
//			FollowLine(i, j, Back, image, orientationField, visited, minutiaes, 
//				section, &sectionAngle, &centerSection, &flag, size, step, partX, partY, &indexOfMinutiae);
//		}
//
//	printf("Finded minutiaes: %d\n	", indexOfMinutiae);
//
//	//printf("%d %d: Lets look i = %d; j = %d \n", blockIdx.x, blockIdx.y, i, j);
//}
//
//bool Start(Minutiae* minutias, float* source, int step, int lengthWings, int width, int height)
//{
//	int sizeSection = lengthWings * 2 + 1;
//
//	CUDAArray<float> image = CUDAArray<float>(source, width, height);
//
//	dim3 blockSize = 1;
//	dim3 gridSize = 1; // dim3(ceilMod(image.Height, defaultThreadCount), ceilMod(image.Width, defaultThreadCount));
//
//	CUDAArray<float> orientationField = CUDAArray<float>(OrientationFieldInPixels(source, width, height), width, height);
//	CUDAArray<bool> visited = CUDAArray<bool>((bool*)calloc(width * height, sizeof(bool)), width, height);
//	CUDAArray<Minutiae> minutiaes = CUDAArray<Minutiae>((Minutiae*)calloc(width * height, sizeof(Minutiae)), width * height, 1);
//
//	FindMinutia << <gridSize, blockSize >> > (image, orientationField, visited, minutiaes, sizeSection, step);
//	cudaDeviceSynchronize();
//	cudaError_t e = cudaGetLastError(); 
//	if (e != cudaSuccess) {
//		printf("Cuda failure %s:%d: '%s'\n", __FILE__, __LINE__, cudaGetErrorString(e));
//		exit(0);
//	}
//
//	//CountOfMinutiaes(countOfMinutiae.GetData(), gridSize.x * gridSize.y);
//
//	//return visited.GetData();
//
//	/*ListOfMinutiae** notProcessedPools = minutiaes.GetData();
//
//	return Parsing(MergeMinutiaePools(notProcessedPools));*/
//
//	minutiaes.GetData(minutias);
//	return visited.GetData();
//}
//
//void outputToFile()
//{
//	freopen("OUTPUT.log", "w", stdout);
//}
//
////int main(int argc, char *argv[])
////{
////	/*DEBUG*/
////	freopen("OUTPUT.log", "w", stdout);
////
////
////	int width;
////	int height;
////	/*if (argc != 2)
////	{
////		printf("Need path to file");
////		return 0;
////	}*/
////	char* filename = "H:\\GitHub\\CUDA-Fingerprinting\\Code\\CUDAFingerprinting.GPU.RidgeLine\\res.bmp";  //Write your way to bmp file
////	int* img = loadBmp(filename, &width, &height);
////	float* source = (float*)malloc(height*width*sizeof(float));
////	for (int i = 0; i < height; i++)
////		for (int j = 0; j < width; j++)
////		{
////			source[i * width + j] = (float)img[i * width + j];
////		}
////
////	Minutiae* foo = (Minutiae*)malloc(sizeof(Minutiae) * width * height);
////
////	bool* res = Start(foo, source, 2, 3, width, height);
////	
////	for (int i = 0; i < height; i++)
////		for (int j = 0; j < width; j++)
////		{
////			img[i * width + j] = res[i * width + j] ? 255 : 0;
////		}
////
////
////
////	saveBmp("..\\rez.bmp", img, width, height);
////
//// 	return 0;
////}
