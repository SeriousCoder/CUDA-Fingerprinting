#include "hip/hip_runtime.h"
#include "kernel.cuh"
#include "hip/hip_runtime.h"
#include <iostream>
#include ""
#include "hip/device_functions.h"
#include <stdio.h>
#include "constsmacros.h"
#include <stdlib.h>
#include <math.h>
#include "ImageLoading.cu"
//#include "CUDAArray.cuh"
#include <float.h>
#include "OrientationField.cu"
#include "Convolution.cu"

__device__ __host__
Point NewPoint(int x, int y)
{
	Point newP;
	newP.x = x;
	newP.y = y;
	return newP;
}

__device__ void AddMinutiae(CUDAArray<Minutiae>* minutiaes, Minutiae minutiae, int* indexOfMinutiae)
{
	minutiaes->SetAt(0, *indexOfMinutiae, minutiae);
	*indexOfMinutiae += 1;
}

__device__ bool OutOfImage(CUDAArray<float> image, int x, int y, int partX, int partY)
{
	//return (x < 0) || (y < 0) || (y >= image.Width) || (x >= image.Height);
	return (x < blockIdx.x * partX) || (y < blockIdx.y * partY) || (x >= (blockIdx.x + 1) * partX) || (y >= (blockIdx.y + 1) * partY) || (y >= image.Height) || (x >= image.Width);
}

__device__
bool OutOfBlock(CUDAArray<float> image, int x, int y, float angle, int partX, int partY)
{
	if ((x < blockIdx.x * partX) || (y < blockIdx.y * partY) || (x >= (blockIdx.x + 1) * partX) || (y >= (blockIdx.y + 1) * partY) || (y >= image.Width) || (x >= image.Height))
	{
		Minutiae newMinutiae;
		newMinutiae.x = x;
		newMinutiae.y = y;
		newMinutiae.angle = angle;
		newMinutiae.type = LineEnding;

		
	}
}

__device__ int NewSection(int x, int y, Direction direction, CUDAArray<float> image, CUDAArray<float> orientationField, 
	Point* section, float* sectionAngle, int* centerSection, bool* flag, int size, int partX, int partY)
{
	int countOfPixels = 1;
	int lengthWings = size / 2;

	for (int i = 0; i < size; i++)
	{
		section[i] = NewPoint(-1, -1);

	}

	int lEnd = lengthWings;
	int rEnd = lEnd;

	bool rightE = false;
	bool leftE = false;

	float angle = -orientationField.At(y, x);
	angle += M_PI_2;

	section[lengthWings] = NewPoint(x, y);

	for (int i = 1; i <= lengthWings; i++)
	{
		int xs = (int)(x - i * cos(angle));
		int ys = (int)(y + i * sin(angle) + 0.95);
		int xe = (int)(x + i * cos(angle) + 0.95);
		int ye = (int)(y - i * sin(angle));

		if (!OutOfImage(image, xs, ys, partX, partY) && (image.At(ys, xs) < 20) && !rightE)
		{
			section[lengthWings - i] = NewPoint(xs, ys);
			rEnd--;
			countOfPixels++;
		}
		else
		{
			rightE = true;
		}

		if (!OutOfImage(image, xe, ye, partX, partY) && (image.At(ye, xe) < 20) && !leftE)
		{
			section[lengthWings + i] = NewPoint(xe, ye);
			lEnd++;
			countOfPixels++;
		}
		else
		{
			leftE = true;
		}

		*centerSection = (lEnd + rEnd) / 2;
	}

	x = section[*centerSection].x;
	y = section[*centerSection].y;

	angle = -orientationField.At(y, x);
	angle += (float) direction * M_PI;
	if (angle < 0) angle += 2.0 * M_PI;

	if (abs(*sectionAngle - angle) > 0.2 && abs(*sectionAngle - angle) < 6) angle += M_PI;
	while (angle > 2.0 * M_PI) angle -= 2 * M_PI;

	*sectionAngle = angle;

	return countOfPixels;
}



__device__ bool CheckAndDeleteFalseMinutia(Minutiae minutia)
{
	return false;
}

__device__ bool MakeStep(int* x, int* y, CUDAArray<float> image, Point* section, int* centerSection, float* sectionAngle, int step, int partX, int partY)
{
	*x = section[*centerSection].x;
	*y = section[*centerSection].y;

	float dx = (float)*x + (float)step * cos(*sectionAngle);
	float dy = (float)*y - (float)step * sin(*sectionAngle);

	*x = (int)(dx >= 0 ? dx + 0.5 : dx - 0.5);
	*y = (int)(dy >= 0 ? dy + 0.5 : dy - 0.5);

	if (OutOfImage(image, *x, *y, partX, partY)) return true;

	return false;
}

__device__ MinutiaeType CheckStopCriteria(int x, int y, CUDAArray<float> image, CUDAArray<bool> visited, int threshold = 20)
{
	if (visited.At(y, x))
		return Intersection;
	if (image.At(y, x) > threshold)
		return LineEnding;

	return NotMinutiae;
}

__device__ void Paint(CUDAArray<float> image, CUDAArray<bool> visited, Point* oldSection, Point* section, int size, int partX, int partY)
{
	Point queue[40];

	int shift = 0; // 30 * (blockIdx.x * gridDim.x + blockIdx.y);
	int rPointer = 0, lPointer = 0;
	//queue = (Point*)malloc(32 * 32 * sizeof(Point));
	Point v1, v2;

	int x1 = -1, x2 = -1, y1 = -1, y2 = -1, x_a, y_a;

	for (int i = 0; i < size; i++)
	{
		if (oldSection[i].x == -1) continue;

		if (x1 == -1)
		{
			x1 = oldSection[i].x;
			y1 = oldSection[i].y;
		}

		x2 = oldSection[i].x;
		y2 = oldSection[i].y;

		visited.SetAt(y2, x2, true);
		queue[shift + rPointer] = oldSection[i];
		rPointer++;
	}

	v1 = NewPoint(x2 - x1, y2 - y1);
	x_a = x1;
	y_a = y1;

	x1 = -1;
	y1 = -1;
	x2 = -1;
	y2 = -1;

	for (int i = 0; i < size; i++)
	{
		if (section[i].x == -1) continue;

		if (x1 == -1)
		{
			x1 = section[i].x;
			y1 = section[i].y;
		}

		x2 = section[i].x;
		y2 = section[i].y;

		visited.SetAt(y2, x2, true);
	}

	v2 = NewPoint(x2 - x1, y2 - y1);

	if (v1.x*v2.x + v1.y*v2.y < 0)
	{
		x1 = x2;
		y1 = y2;
		v1.x = -v1.x;
		v1.y = -v1.y;
	}

	while (abs(rPointer - lPointer) > 0)
	{
		for (int i = -2; i < 3; i++)
			for (int j = -2; j < 3; j++)
			{
				if (i == 0 && j == 0) continue;

				int x = queue[lPointer].x + i;
				int y = queue[lPointer].y + j;

				if (OutOfImage(image, x, y, partX, partY) || visited.At(y, x) || image.At(y, x) > 20) continue;

				Point pointV1 = NewPoint(x_a - x, y_a - y);
				Point pointV2 = NewPoint(x1 - x, y1 - y);

				int skew1 = v1.x*pointV1.y - pointV1.x*v1.y >= 0 ? 1 : -1;
				int skew2 = v2.x*pointV2.y - pointV2.x*v2.y >= 0 ? 1 : -1;

				if (skew1*skew2 < 0)
				{
					queue[shift + rPointer] = NewPoint(x, y);
					rPointer++;
					visited.SetAt(y, x, true);
					if (rPointer == 40) rPointer = 0;
				}
			}

		lPointer++;
		if (lPointer == 40) lPointer = 0;
	}
}

__device__
void Paint2(CUDAArray<float> image, CUDAArray<bool> visited, Point* oldSection, float* sectionAngle, int size, int partX, int partY)
{
	Point queue[40];

	int rPointer = 0, lPointer = 0;

	for (int i = 0; i < size; i++)
	{
		visited.SetAt(oldSection[i].y, oldSection[i].x, true);
		queue[rPointer] = oldSection[i];
		rPointer++;
	}

	while (abs(rPointer - lPointer) > 0)
	{
		int x = queue[lPointer].x, y = queue[lPointer].y;

		Point foo = NewPoint(x, y);
		int bar = 0;

		MakeStep(&x, &y, image, &foo, &bar, sectionAngle, 1, partX, partY);

		for (int i = -1; i < 2; i++)
			for (int j = -1; j < 2; j++)
			{
				if (OutOfImage(image, x + i, y + j, partX, partY)) continue;
				if (image.At(y + j, x + i) < 20 && !visited.At(y + j, x + i))
				{
					visited.SetAt(y + j, x + i, true);
					queue[rPointer] = NewPoint(x + i, y + j);
					rPointer++;
					if (rPointer == 40) rPointer = 0;
				}
			}

		lPointer++;
		if (lPointer == 40) lPointer = 0;
	}
}

__device__ void FollowLine(int x, int y, Direction direction, CUDAArray<float> image, CUDAArray<float> orientationField,
	CUDAArray<bool> visited, CUDAArray<Minutiae> minutiaes,	Point* section, float* sectionAngle, 
	int* centerSection, bool* flag, int size, int step, int partX, int partY, int* indexOfMinutiae)
{
	int countOfPixels = NewSection(x, y, direction, image, orientationField, section, sectionAngle, 
		centerSection, flag, size, partX, partY);
	if (countOfPixels < 3) return;

	MinutiaeType type;

	Point* oldSection = new Point[size];
	bool outOfBlock = false;

	do
	{		
		for (int i = 0; i < size; i++){
			oldSection[i] = section[i];
		}

		outOfBlock = MakeStep(&x, &y, image, section, centerSection, sectionAngle, step, partX, partY);
		
		if (outOfBlock)
		{
			Paint2(image, visited, oldSection, sectionAngle, size, partX, partY);
			return;
		}

		type = CheckStopCriteria(x, y, image, visited);

		NewSection(x, y, direction, image, orientationField, section, sectionAngle, centerSection, flag, size, partX, partY);
		if (section[*centerSection].x == -1) return;

		Paint(image, visited, oldSection, section, size, partX, partY);
	} while (type == NotMinutiae);

	Minutiae possMinutiae;
	possMinutiae.x = x;
	possMinutiae.y = y;
	possMinutiae.angle = *sectionAngle;
	possMinutiae.type = type;

	AddMinutiae(&minutiaes, possMinutiae, indexOfMinutiae);
}

__global__ void FindMinutia(CUDAArray<float> image, CUDAArray<float> orientationField, CUDAArray<bool> visited,
	CUDAArray<Minutiae> minutiaes, const int size, const int step, int colorThreshold = 15)
{
	Point* section = new Point[size];
	float sectionAngle;
	int centerSection;
	bool flag  = false;

	int partX = 32; //image.Height / gridDim.x;
	int partY = 32; //image.Width / gridDim.y;

	int indexOfMinutiae = blockIdx.x * image.Height + blockIdx.y * defaultThreadCount;

	//printf("%d %d %d\n", blockIdx.x, blockIdx.y, threadIdx.x);

	//if (blockIdx.x == 5 && blockIdx.y == 10)
	for (int i = blockIdx.x * partX; i < (blockIdx.x + 1) * partX; i++)
		for (int j = blockIdx.y * partY; j < (blockIdx.y + 1) * partY; j++)
	//for (int i = 0; i < image.Width; i++)
	//	for (int j = 0; j < image.Height; j++)
		{
			if (OutOfImage(image, i, j, partX, partY))
			{
				//printf("Tu-tu. %d %d\n", i, j);
				continue;
			}

			if ((image.At(j, i) >= colorThreshold) || visited.At(j, i)) continue;
			
			sectionAngle = -orientationField.At(j, i);
			if (sectionAngle < 0) sectionAngle += 2.0 * M_PI;
			FollowLine(i, j, Forward, image, orientationField, visited, minutiaes, 
				section, &sectionAngle, &centerSection, &flag, size, step, partX, partY, &indexOfMinutiae);

			flag = false;

			sectionAngle = -orientationField.At(j, i) + M_PI;
			FollowLine(i, j, Back, image, orientationField, visited, minutiaes, 
				section, &sectionAngle, &centerSection, &flag, size, step, partX, partY, &indexOfMinutiae);
		}

	//printf("Finded minutiaes: %d\n	", indexOfMinutiae);

	//printf("%d %d: Lets look i = %d; j = %d \n", blockIdx.x, blockIdx.y, i, j);
}

void saveMyBmp(bool* visited, int width, int height)
{
	int* img = (int*)malloc(width * height * sizeof(int));

	for (int i = 0; i < height; i++)
		for (int j = 0; j < width; j++)
		{
			if (visited[i * width + j]) img[i * width + j] = 255; else img[i * width + j] = 0;
		}

	char filename[80];
	sprintf(filename, "resGPU.bmp");

	saveBmp(filename, img, width, height);
}

bool Start(Minutiae* minutias, float* source, int step, int lengthWings, int width, int height)
{
	int sizeSection = lengthWings * 2 + 1;

	CUDAArray<float> image = CUDAArray<float>(source, width, height);

	dim3 blockSize = 1;
	dim3 gridSize = dim3(ceilMod(image.Width, defaultThreadCount), ceilMod(image.Height, defaultThreadCount));

	printf("GridDim: %d %d\n", gridSize.x, gridSize.y);

	CUDAArray<float> orientationField = CUDAArray<float>(OrientationFieldInPixels(source, width, height), width, height);
	CUDAArray<bool> visited = CUDAArray<bool>((bool*)calloc(width * height, sizeof(bool)), width, height);
	CUDAArray<Minutiae> minutiaes = CUDAArray<Minutiae>((Minutiae*)calloc(width * height, sizeof(Minutiae)), width * height, 1);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
	FindMinutia << <gridSize, blockSize >> > (image, orientationField, visited, minutiaes, sizeSection, step);
	hipEventRecord(stop);

	hipDeviceSynchronize();
	hipError_t e = hipGetLastError(); 
	if (e != hipSuccess) {
		printf("Cuda failure %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(e));
		//exit(0);
	}

	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	printf("Time: %.2f\n", milliseconds);

	//CountOfMinutiaes(countOfMinutiae.GetData(), gridSize.x * gridSize.y);

	//return visited.GetData();

	/*ListOfMinutiae** notProcessedPools = minutiaes.GetData();

	return Parsing(MergeMinutiaePools(notProcessedPools));*/

	minutiaes.GetData(minutias);

	//printf("Starting search a duplications\n");
	DeleteDuplicate(minutias, height * width);

	//saveMyBmp(visited.GetData(), width, height);

	return true;
}

void outputToFile()
{
	freopen("OUTPUT.log", "w", stdout);
}

//int main(int argc, char *argv[])
//{
//	/*DEBUG*/
//	freopen("OUTPUT.log", "w", stdout);
//
//
//	int width;
//	int height;
//	/*if (argc != 2)
//	{
//		printf("Need path to file");
//		return 0;
//	}*/
//	char* filename = "H:\\GitHub\\CUDA-Fingerprinting\\Code\\CUDAFingerprinting.GPU.RidgeLine\\res.bmp";  //Write your way to bmp file
//	int* img = loadBmp(filename, &width, &height);
//	float* source = (float*)malloc(height*width*sizeof(float));
//	for (int i = 0; i < height; i++)
//		for (int j = 0; j < width; j++)
//		{
//			source[i * width + j] = (float)img[i * width + j];
//		}
//
//	Minutiae* foo = (Minutiae*)malloc(sizeof(Minutiae) * width * height);
//
//	bool* res = Start(foo, source, 2, 3, width, height);
//	
//	for (int i = 0; i < height; i++)
//		for (int j = 0; j < width; j++)
//		{
//			img[i * width + j] = res[i * width + j] ? 255 : 0;
//		}
//
//
//
//	saveBmp("..\\rez.bmp", img, width, height);
//
// 	return 0;
//}
