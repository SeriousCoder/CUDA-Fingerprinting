#include "hip/hip_runtime.h"
#include "kernel.cuh"
#define M_PI 3.14159265358979323846
#define Pi4 (M_PI / 4);

class RidgeOnLine
{
	
private:
	int _step;
	int _sizeSection;
	int _height;
	int _width;

public:
	RidgeOnLine()
	{
		
	}

	~RidgeOnLine()
	{

	}
};

__device__ __host__
Point NewPoint(int x, int y)
{
	Point newP;
	newP.x = x;
	newP.y = y;
	return newP;
}

__device__ void AddMinutiae(CUDAArray<int> countOfMinutiae, CUDAArray<ListOfMinutiae*> minutiaes, Minutiae minutiae)
{
	minutiaes.At(blockIdx.x, 0)->Add(minutiae);
	countOfMinutiae.SetAt(blockIdx.x, 0, countOfMinutiae.At(blockIdx.x, 0) + 1);
}

//make for blocks
__device__ bool OutOfImage(CUDAArray<float> image, int x, int y)
{
	return (x < 0) || (y < 0) || (x >= image.Width) || (y >= image.Height);
}

__device__ void NewSection(int sx, int sy, Direction direction, CUDAArray<float> image, CUDAArray<float> orientationField, 
	Point* section,	float* sectionAngle, int* centerSection, bool* flag, int size)
{
	int lengthWings = size / 2;

	for (int i = 0; i < size; i++)
	{
		section[i] = NewPoint(-1, -1);

	}

	int x = sx;
	int y = sy;

	int lEnd = lengthWings;
	int rEnd = lEnd;

	bool rightE = false;
	bool leftE = false;

	float angle = orientationField.At(x, y) + M_PI / 2;

	for (int i = 1; i <= lengthWings; i++)
	{
		int xs = (int)(x - i * cos(angle));
		int ys = (int)(y - i * sin(angle));
		int xe = (int)(x + i * cos(angle));
		int ye = (int)(y + i * sin(angle));

		if (!OutOfImage(image, xs, ys) && (image.At(xs, ys)) && !rightE)
		{
			section[lengthWings - i] = NewPoint(xs, ys);
			rEnd--;
		}
		else
		{
			rightE = true;
		}

		if (!OutOfImage(image, xe, ye) && (image.At(xe, ye)) && !leftE)
		{
			section[lengthWings - i] = NewPoint(xe, ye);
			lEnd--;
		}
		else
		{
			leftE = true;
		}

		*centerSection = (lEnd + rEnd) / 2;

		x = section[*centerSection].x;
		y = section[*centerSection].y;
	}

	angle = orientationField.At(x, y) + direction * M_PI;
	if (angle < 0) angle += 2 * M_PI;

	if (*flag){
		if (abs(*sectionAngle - angle) > 0.2 && abs(*sectionAngle - angle) < 6) angle + M_PI;
	}
	else *flag = true;

	*sectionAngle = angle;
}



__device__ bool CheckAndDeleteFalseMinutia(Minutiae minutia)
{
	return true;
}

__device__ Point MakeStep(CUDAArray<float> image, Point* section, int* centerSection, float* sectionAngle, int step)
{
	int x = section[*centerSection].x;
	int y = section[*centerSection].y;

	float dx = (float)x + (float)step * cos(*sectionAngle);
	float dy = (float)y + (float)step * sin(*sectionAngle);

	x = (int)(dx >= 0 ? dx + 0.5 : dx - 0.5);
	y = (int)(dy >= 0 ? dy + 0.5 : dy - 0.5);

	return OutOfImage(image, x, y) ? NewPoint(-1, -1) : NewPoint(x, y);
}

__device__ MinutiaeType CheckStopCriteria(CUDAArray<float> image, CUDAArray<bool> visited, Point* section, int* centerSection, int threshold = 20)
{
	int x = section[*centerSection].x;
	int y = section[*centerSection].y;

	if (visited.At(x, y))
		return Intersection;
	if (image.At(x, y) > threshold)
		return LineEnding;

	return NotMinutiae;
}

__device__ void Paint(CUDAArray<float> image, CUDAArray<bool> visited, Point* oldSection, Point* section, int size)
{
	Queue* queue = new Queue;
	Point v1, v2;

	int x1 = -1, x2 = -1, y1 = -1, y2 = -1, x_a, y_a;

	for (int i = 0; i < size; i++)
	{
		if (oldSection[i].x == -1) continue;

		if (x1 == -1)
		{
			x1 = oldSection[i].x;
			y1 = oldSection[i].y;
		}

		x2 = oldSection[i].x;
		y2 = oldSection[i].y;

		visited.SetAt(oldSection[i].x, oldSection[i].y, true);
		queue->Push(oldSection[i]);
	}

	v1 = NewPoint(x2 - x1, y2 - y1);
	x_a = x1;
	y_a = y1;

	x1 = -1;
	y1 = -1;
	x2 = -1;
	y2 = -1;

	for (int i = 0; i < size; i++)
	{
		if (section[i].x == -1) continue;

		if (x1 == -1)
		{
			x1 = section[i].x;
			y1 = section[i].y;
		}

		x2 = section[i].x;
		y2 = section[i].y;

		visited.SetAt(section[i].x, section[i].y, true);
		queue->Push(section[i]);
	}

	v2 = NewPoint(x2 - x1, y2 - y1);

	if (v1.x*v2.x + v1.y*v2.y < 0)
	{
		x1 = x2;
		y1 = y2;
		v1 = NewPoint(-v1.x, -v1.y);
	}

	while (queue->count > 0)
	{
		Point point = queue->Pop();

		int cX = point.x;
		int cY = point.y;

		for (int i = -1; i < 2; i++)
			for (int j = -1; j < 2; j++)
			{
				if (i == 0 && j == 0) continue;

				int x = cX + i;
				int y = cY + j;

				if (OutOfImage(image, x, y) || visited.At(x, y) || image.At(x, y) > 15) continue;

				Point pointV1 = NewPoint(x_a - x, y_a - y);
				Point pointV2 = NewPoint(x1 - x, y1 - y);

				int skew1 = v1.x*pointV1.y - pointV1.x*v1.y >= 0 ? 1 : -1;
				int skew2 = v2.x*pointV2.y - pointV2.x*v2.y >= 0 ? 1 : -1;

				if (skew1*skew2 < 0)
				{
					queue->Push(NewPoint(x, y));
					visited.SetAt(x, y, true);
				}
			}
	}
}

__device__ void FollowLine(int x, int y, Direction direction, CUDAArray<float> image, CUDAArray<float> orientationField,
	CUDAArray<bool> visited, CUDAArray<int> countOfMinutiae, CUDAArray<ListOfMinutiae*> minutiaes,
	Point* section, float* sectionAngle, int* centerSection, bool* flag, int size, int step)
{
	NewSection(x, y, direction, image, orientationField, section, sectionAngle, centerSection, flag, size);
	if (section[*centerSection].x == -1) return;

	MinutiaeType type;
	Point point;

	do
	{
		Point* oldSection = (Point*)malloc(sizeof(Point) * size);
		for (int i = 0; i < size; i++)
			oldSection[i] = section[i];

		point = MakeStep(image, section, centerSection, sectionAngle, step);

		if (point.x == -1) return;

		NewSection(point.x, point.y, direction, image, orientationField, section, sectionAngle, centerSection, flag, size);
		if (section[*centerSection].x == -1) return;

		type = CheckStopCriteria(image, visited, section, centerSection);

		Paint(image, visited, oldSection, section, size);
	} while (type == NotMinutiae);

	Minutiae possMinutiae;
	possMinutiae.x = point.x;
	possMinutiae.y = point.y;
	possMinutiae.angle = *sectionAngle;
	possMinutiae.type = type;

	//if (IsDuplicate(possMinutiae)) return;

	if (!CheckAndDeleteFalseMinutia(possMinutiae))
	{
		AddMinutiae(countOfMinutiae, minutiaes, possMinutiae);
	}
}

__global__ void FindMinutia(CUDAArray<float> image, CUDAArray<float> orientationField, CUDAArray<bool> visited,
	CUDAArray<int> countOfMinutiae, CUDAArray<ListOfMinutiae*> minutiaes, 
	const int size, const int step, int colorThreshold = 15)
{
	Point* section = new Point[size];
	float sectionAngle;
	int centerSection;
	bool flag;
	minutiaes.SetAt(blockIdx.x, 0, new ListOfMinutiae);

	for (int i = blockIdx.x * gridDim.x; i < (blockIdx.x + 1) * gridDim.x; i++)
		for (int j = blockIdx.y * gridDim.x; j < (blockIdx.y + 1) * gridDim.x; j++)
		{
			if ((image.At(i, j) >= colorThreshold) || visited.At(i, j)) return;
			visited.SetAt(i, j, true);

			FollowLine(i, j, Forward, image, orientationField, visited, countOfMinutiae, minutiaes, 
				section, &sectionAngle, &centerSection, &flag, size, step);
			FollowLine(i, j, Back, image, orientationField, visited, countOfMinutiae, minutiaes, 
				section, &sectionAngle, &centerSection, &flag, size, step);
		}
}

bool* Start(float* source, int step, int lengthWings, int width, int height)
{
	int sizeSection = lengthWings * 2 + 1;

	CUDAArray<float> image = CUDAArray<float>(source, width, height);
	CUDAArray<float> orientationField = CUDAArray<float>(OrientationFieldInBlocks(source, width, height), height, width);
	CUDAArray<bool> visited = CUDAArray<bool>((bool*)calloc(width * height, sizeof(bool)), width, height);
	CUDAArray<int> countOfMinutiae = CUDAArray<int>((int*)calloc(defaultThreadCount, sizeof(int)), defaultThreadCount, 1);
	CUDAArray<ListOfMinutiae*> minutiaes = CUDAArray<ListOfMinutiae*>((ListOfMinutiae**)calloc(defaultThreadCount, sizeof(ListOfMinutiae*)), defaultThreadCount, 1);

	dim3 blockSize = 1;
	dim3 gridSize = dim3(defaultThreadCount);

	FindMinutia << <gridSize, blockSize >> > (image, orientationField, visited, countOfMinutiae, minutiaes, sizeSection, step);

	return visited.GetData();
}

int main()
{
	int width;
	int height;
	char* filename = "H:\\GitHub\\CUDA-Fingerprinting\\Code\\CUDAFingerprinting.GPU.RidgeLine\\res.bmp";  //Write your way to bmp file
	int* img = loadBmp(filename, &width, &height);
	float* source = (float*)malloc(height*width*sizeof(float));
	for (int i = 0; i < height; i++)
		for (int j = 0; j < width; j++)
		{
			source[i * width + j] = (float)img[i * width + j];
		}

	bool* res = Start(source, 2, 3, width, height);
	
	for (int i = 0; i < height; i++)
		for (int j = 0; j < width; j++)
		{
			img[i * width + j] = res[i * width + j] ? 0 : 255;
		}

	saveBmp("..\\rez.bmp", img, width, height);

	return 0;
}