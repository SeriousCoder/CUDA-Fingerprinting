#include "hip/hip_runtime.h"
#include "kernel.cuh"
#include "hip/hip_runtime.h"
#include <math.h>
#include <stdlib.h>
#include <stdio.h>

void Sort(Minutiae* minutiaes, int size);
void merge(Minutiae* minutiaes, int left, int mid, int right);

void DeleteDuplicate(Minutiae* minutiaes, int size, int delta)
{
	//printf("Starting search a duplications\n");
	Sort(minutiaes, size);

	for (int i = 1; i < size; i++)
	{
		if (minutiaes[i].type == 0) break;

		for (int j = 0; j < i; j++)
		{
			if (minutiaes[i].type == minutiaes[j].type)
				if (sqrt(pow((float)minutiaes[i].x - minutiaes[j].x, 2) +
					pow((float)minutiaes[i].y - minutiaes[j].y, 2)) < delta)
				{
					minutiaes[i].type = 0;
					break;
				}
		}
	}
}

void Sort(Minutiae* minutiaes, int size)
{
	for (int i = 1; i < size; i *= 2)
		for (int j = 0; j < size - i; j += 2 * i)
			merge(minutiaes, j, j + i, min(j + 2 * i, size));
}

void merge(Minutiae* minutiaes, int left, int mid, int right)
{
	int it1 = 0, it2 = 0;
	Minutiae* result = (Minutiae*)malloc((right - left) * sizeof(Minutiae));

	while (left + it1 < mid && mid + it2 < right)
	{
		if (minutiaes[left + it1].type == minutiaes[mid + it2].type &&
			minutiaes[mid + it2].type == NotMinutiae) break;

		if (minutiaes[left + it1].type == minutiaes[mid + it2].type)
		{
			if (minutiaes[left + it1].x * 1000 + minutiaes[left + it1].y <
				minutiaes[mid + it2].x * 1000 + minutiaes[mid + it2].y)
			{
				result[it1 + it2] = minutiaes[left + it1];
				it1++;
			}
			else
			{
				result[it1 + it2] = minutiaes[mid + it2];
				it2++;
			}
		}
		else if (minutiaes[left + it1].type != NotMinutiae)
		{
			result[it1 + it2] = minutiaes[left + it1];
			it1++;
		}
		else
		{
			result[it1 + it2] = minutiaes[mid + it2];
			it2++;
		}

	}

	while (left + it1 < mid)
	{
		result[it1 + it2] = minutiaes[left + it1];
		it1++;
	}

	while (mid + it2 < right)
	{
		result[it1 + it2] = minutiaes[mid + it2];
		it2++;
	}

	for (int i = 0; i < it1 + it2; i++)
		minutiaes[left + i] = result[i];

	free(result);
}