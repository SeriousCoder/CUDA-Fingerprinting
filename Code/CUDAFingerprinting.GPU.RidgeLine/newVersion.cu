
#include <hip/hip_runtime.h>
//#include "kernel.cuh"
//#include "cuda_runtime.h"
//#include <iostream>
//#include "device_launch_parameters.h"
//#include "device_functions.h"
//#include <stdio.h>
//#include "constsmacros.h"
//#include <stdlib.h>
//#include <math.h>
//#include "ImageLoading.cu"
////#include "CUDAArray.cuh"
//#include <float.h>
//#include "OrientationField.cu"
//#include "Convolution.cu"
//
//#define M_PI 3.14159265358979323846
//
//Point NewPoint(int x, int y)
//{
//	Point newP;
//	newP.x = x;
//	newP.y = y;
//	return newP;
//}
//
//int countOfEndings;
//
//void AddMinutiae(Minutiae* minutiaes, Minutiae minutiae, int* indexOfMinutiae)
//{
//	minutiaes[*indexOfMinutiae] = minutiae;
//	(*indexOfMinutiae)++;
//	printf("Added new minutia. Type = %d\n", minutiae.type);
//	if (minutiae.type == 1) countOfEndings++;
//}
//
//bool OutOfImage(int x, int y, float* image, int* width, int* height)
//{
//	return (x < 0) || (y < 0) || (x >= *width) || (y >= *height);
//}
//
//void NewSection(int x, int y, Direction direction, float* image, float* orientField,
//	Point* section, float* sectionAngle, int* centerSection, bool* flag,
//	int sizeOfSection, int* width, int* height)
//{
//	printf("=+=+=+=+=+=+=+=+=+=\n");
//	printf("Making new section from point (%d, %d)\n", x, y);
//
//	for (int i = 0; i < sizeOfSection; i++)
//	{
//		section[i].x = -1;
//		section[i].y = -1;
//	}
//
//	int wing = sizeOfSection / 2;
//
//	int lEnd = wing, rEnd = lEnd;
//	bool rightE = false, leftE = false;
//
//	float angle = -orientField[y * (*width) + x];
//	printf("Angle in this pixel: %f\n", angle);
//	angle += (float)M_PI_2;
//
//	section[wing] = NewPoint(x, y);
//
//	for (int i = 1; i <= wing; i++)
//	{
//		int xs = (int)(x - i * cos(angle));
//		int ys = (int)(y + i * sin(angle) + 0.95);
//		int xe = (int)(x + i * cos(angle) + 0.95);
//		int ye = (int)(y - i * sin(angle));
//
//		//printf("Left pixel = (%d, %d); right pixel = (%d, %d)\n", xs, ys, xe, ye);
//
//		if (!OutOfImage(xs, ys, image, width, height) && (image[ys * (*width) + xs] < 20) && !rightE)
//		{
//			section[wing - i] = NewPoint(xs, ys);
//			rEnd--;
//		}
//		else
//		{
//			rightE = true;
//		}
//
//		if (!OutOfImage(xe, ye, image, width, height) && (image[ye * (*width) + xe] < 20) && !leftE)
//		{
//			section[wing + i] = NewPoint(xe, ye);
//			lEnd++;
//		}
//		else
//		{
//			leftE = true;
//		}
//
//		*centerSection = (lEnd + rEnd) / 2;
//	}
//	
//	x = section[*centerSection].x;
//	y = section[*centerSection].y;
//
//	printf("New center: (%d, %d)\n", x, y);
//
//	angle = -orientField[y * (*width) + x];
//	angle += (float) direction * M_PI;
//	if (angle < 0) angle += 2.0 * M_PI;
//
//	if (abs(*sectionAngle - angle) > 0.2 && abs(*sectionAngle - angle) < 6) angle += M_PI;
//	while (angle > 2.0 * M_PI) angle -= 2 * M_PI;
//
//	printf("Old angle: %f\nNew angle: %f\n", *sectionAngle, angle);
//
//	*sectionAngle = angle;
//
//	printf("=+=+=+=+=+=+=+=+=+=\n");
//}
//
//void MakeStep(int* x, int* y, float* image, float* sectionAngle, Point* section,
//	int* sectionCenter, int step, int* width, int* height)
//{
//	*x = section[*sectionCenter].x;
//	*y = section[*sectionCenter].y;
//
//	float dx = (float)*x + step * cos(*sectionAngle);
//	float dy = (float)*y - step * sin(*sectionAngle);
//
//	*x = (int)(dx >= 0 ? dx + 0.5 : dx - 0.5);
//	*y = (int)(dy >= 0 ? dy + 0.5 : dy - 0.5);
//
//	if (OutOfImage(*x, *y, image, width, height))
//	{
//		*x = -1;
//		*y = -1;
//	}
//}
//
//int steps;
//
//void Paint(float* image, bool* visited, Point* oldSection, Point* newSection,
//	int size, int* width, int* height)
//{
//	Point queue[400];
//	Point v1, v2;
//
//	int left = 0, right = 0;
//
//	int x1 = -1, x2 = -1, y1 = -1, y2 = -1, x_a, y_a;
//
//	for (int i = 0; i < size; i++)
//	{
//		if (oldSection[i].x == -1) continue;
//
//		if (x1 == -1)
//		{
//			x1 = oldSection[i].x;
//			y1 = oldSection[i].y;
//		}
//
//		x2 = oldSection[i].x;
//		y2 = oldSection[i].y;
//
//		visited[y2 * (*width) + x2] = true;
//		queue[right] = NewPoint(x2, y2);
//		right++;
//	}
//
//	v1 = NewPoint(x2 - x1, y2 - y1);
//	x_a = x1; 
//	y_a = y1;
//
//	x1 = -1;
//	y1 = -1;
//	x2 = -1;
//	y2 = -1;
//
//	for (int i = 0; i < size; i++)
//	{
//		if (newSection[i].x == -1) continue;
//
//		if (x1 == -1)
//		{
//			x1 = newSection[i].x;
//			y1 = newSection[i].y;
//		}
//
//		x2 = newSection[i].x;
//		y2 = newSection[i].y;
//
//		visited[y2 * (*width) + x2] = true;
//	}
//
//	v2 = NewPoint(x2 - x1, y2 - y1);
//
//	if (v1.x*v2.x + v1.y*v2.y < 0)
//	{
//		x1 = x2;
//		y1 = y2;
//		v1.x = -v1.x;
//		v1.y = -v1.y;
//	}
//
//	while (abs(right - left) > 0)
//	{
//		for (int i = -2; i < 3; i++)
//			for (int j = -2; j < 3; j++)
//			{
//				if (i == 0 && j == 0) continue;
//
//				int x = queue[left].x + i;
//				int y = queue[left].y + j;
//
//				if (OutOfImage(x, y, image, width, height) || visited[y * (*width) + x] || image[y * (*width) + x] > 20) continue;
//
//				Point pointV1 = NewPoint(x_a - x, y_a - y);
//				Point pointV2 = NewPoint(x1 - x, y1 - y);
//
//				int skew1 = v1.x*pointV1.y - pointV1.x * v1.y >= 0 ? 1 : -1;
//				int skew2 = v2.x*pointV2.y - pointV2.x * v2.y >= 0 ? 1 : -1;
//
//				if (skew1*skew2 < 0)
//				{
//					queue[right] = NewPoint(x, y);
//					visited[y * (*width) + x] = true;
//					right++;
//					if (right == 400) right = 0;
//				}
//			} 
//
//		left++;
//		if (left == 400) left = 0;
//	}
//}
//
//MinutiaeType CheckStopCriteria(int x, int y, float* image, bool* visited, int* width, int threshold = 20)
//{
//	if (visited[y * (*width) + x])
//		return Intersection;
//
//	if (image[y * (*width) + x] > threshold)
//		return LineEnding;
//
//	return NotMinutiae;
//}
//
//void saveMyBmp_Paint(bool* visited, int index, int width, int height)
//{
//	int* img = (int*)malloc(width * height * sizeof(int));
//
//	for (int i = 0; i < height; i++)
//		for (int j = 0; j < width; j++)
//		{
//			if (visited[i * width + j]) img[i * width + j] = 255; else img[i * width + j] = 0;
//		}
//
//	char filename[80];
//	sprintf(filename, "res%d-%d.bmp", index, steps);
//
//	steps++;
//
//	saveBmp(filename, img, width, height);
//}
//
//void saveMyBmp(bool* visited, int* index, int width, int height)
//{
//	int* img = (int*)malloc(width * height * sizeof(int));
//
//	for (int i = 0; i < height; i++)
//		for (int j = 0; j < width; j++)
//		{
//			if (visited[i * width + j]) img[i * width + j] = 255; else img[i * width + j] = 0;
//		}
//
//	char filename[80];
//	sprintf(filename, "res%d.bmp", *index);
//
//	saveBmp(filename, img, width, height);
//}
//
//
//void FollowLine(int x, int y, Direction direction, float* image, float* orientField,
//	bool* visited, Minutiae* minutiaes, Point* section, float* sectionAngle,
//	int* centerSection, bool* flag, int* sizeOfSection, int* step, int* width, 
//	int* height, int* indexOfMinutiae)
//{
//	NewSection(x, y, direction, image, orientField, section,
//		sectionAngle, centerSection, flag, *sizeOfSection, width, height);
//	if (section[*centerSection].x == -1) return;
//
//	MinutiaeType type; steps = 0;
//	//int x, y;
//
//	Point* oldSection = new Point[*sizeOfSection];
//
//	do
//	{
//		printf("First/last section: \n");
//		for (int i = 0; i < *sizeOfSection; i++)
//		{
//			oldSection[i] = section[i];
//			printf("(%d, %d) ", oldSection[i].x, oldSection[i].y);
//		}
//
//		printf("\n");
//
//		MakeStep(&x, &y, image, sectionAngle, section, centerSection,
//			*step, width, height); 
//
//		printf("Step complited. New point: (%d, %d)\n", x, y);
//
//		if (x == -1) return;
//
//		NewSection(x, y, direction, image, orientField, section,
//			sectionAngle, centerSection, flag, *sizeOfSection, width, height);
//		if (section[*centerSection].x == -1) return;
//
//		type = CheckStopCriteria(x, y, image, visited, width);
//
//		Paint(image, visited, oldSection, section, *sizeOfSection, 
//			width, height);
//		//saveMyBmp_Paint(visited, *indexOfMinutiae, *width, *height);
//	} while (type == NotMinutiae);
//
//	Minutiae newMinutiae;
//	newMinutiae.x = x;
//	newMinutiae.y = y;
//	newMinutiae.angle = *sectionAngle;
//	newMinutiae.type = type;
//
//	
//
//	AddMinutiae(minutiaes, newMinutiae, indexOfMinutiae);
//	
//	/*if (type == LineEnding) *///saveMyBmp(visited, indexOfMinutiae, *width, *height);
//}
//
//void FindMinutiae(float* image, float* orientField, bool* visited,
//	Minutiae* minutiaes, int sizeOfSection, int* width, 
//	int* height, int step, int colorThreshold = 20)
//{
//	Point* section = new Point[sizeOfSection];
//	float sectionAngle;
//	int centerSection;
//	bool flag = false;
//
//	int indexOfMinutiae = 0;
//
//	for (int i = 0; i < *width; i++)
//		for (int j = 0; j < *height; j++)
//		{
//			//On parallel version need add check out of image
//			if ((image[j * (*width) + i] >= colorThreshold) || (visited[j * (*width) + i]))
//				continue;
//
//			printf("================================================================\n");
//			printf("Starting point: (%d, %d)  //color = %f\n", i, j, image[j * (*width) + i]);
//			printf("     ***** Minutiae #%d *****     \n", indexOfMinutiae);
//			printf("Forward....\n");
//
//			sectionAngle = -orientField[j * (*width) + i];
//			if (sectionAngle < 0) sectionAngle += 2.0 * M_PI;
//			FollowLine(i, j, Forward, image, orientField, visited, minutiaes,
//				section, &sectionAngle, &centerSection, &flag, &sizeOfSection,
//				&step, width, height, &indexOfMinutiae);
//
//			flag = false;
//
//			printf("-----------------------------\n");
//			printf("     ***** Minutiae #%d *****     \n", indexOfMinutiae);
//			printf("Back....\n");
//			sectionAngle = -orientField[j * (*width) + i] + M_PI;
//			//if (sectionAngle > 2.0 * M_PI) sectionAngle -= M_PI;
//			FollowLine(i, j, Back, image, orientField, visited, minutiaes,
//				section, &sectionAngle, &centerSection, &flag, &sizeOfSection,
//				&step, width, height, &indexOfMinutiae);
//		}
//
//	printf("Finded minutiaes: %d\n", indexOfMinutiae);
//}
//
//bool Start(Minutiae* minutiaeOut, float* source, int step, int lengthWings, int width, int height)
//{
//	/*DEBUG*/
//	freopen("OUTPUT.log", "w", stdout);
//
//	countOfEndings = 0;
//
//	bool* visited = (bool*)calloc(width * height, sizeof(bool));
//	int countOfMinutiae = 0;
//	Minutiae* minutiaes = (Minutiae*)calloc(width * height, sizeof(Minutiae));
//
//	float* orientFieldin = OrientationFieldInPixels(source, width, height);
//
//	/*printf("Angles:\n");
//	for (int i = 0; i < width; i++) {
//		for (int j = 0; j < height; j++)
//		{
//			if (source[j * width + i] < 15)
//				printf("%f ", orientFieldin[j * (width)+i]);
//			else
//				printf("0.000000 ", orientFieldin[j * (width)+i]);
//		}
//		printf("\n");
//	}*/
//
//	FindMinutiae(source, orientFieldin, visited, minutiaeOut, lengthWings * 2 + 1, &width, &height, step);
//
//	//minutiaeOut = minutiaes;
//
//	printf("Endings: %d", countOfEndings);
//
//	return false;
//}
//
////int main(int argc, char *argv[])
////{
////	/*DEBUG*/
////	freopen("OUTPUT.log", "w", stdout);
////
////
////	int width;
////	int height;
////	/*if (argc != 2)
////	{
////	printf("Need path to file");
////	return 0;
////	}*/
////	char* filename = "H:\\GitHub\\CUDA-Fingerprinting\\Code\\CUDAFingerprinting.GPU.RidgeLine\\res.bmp";  //Write your way to bmp file
////	int* img = loadBmp(filename, &width, &height);
////	float* source = (float*)malloc(height*width*sizeof(float));
////	for (int i = 0; i < height; i++)
////		for (int j = 0; j < width; j++)
////		{
////			source[i * width + j] = (float)img[i * width + j];
////		}
////
////	//Minutiae* foo = (Minutiae*)malloc(sizeof(Minutiae) * width * height);
////
////	float* orientField = OrientationFieldInPixels(source, width, height);
////
////	/*for (int i = 0; i < width; i++)
////		printf("%f\n", orientField[30 * (width) + i]);*/
////
////	Start(source, orientField, 2, 3, width, height);
////
////	/*for (int i = 0; i < height; i++)
////	for (int j = 0; j < width; j++)
////	{
////	img[i * width + j] = res[i * width + j] ? 255 : 0;
////	}
////*/
////
////
////	//saveBmp("..\\rez.bmp", img, width, height);
////
////	return 0;
////}