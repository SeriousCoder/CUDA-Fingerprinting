
#include <hip/hip_runtime.h>
//#include "kernel.cuh"
//#include "cuda_runtime.h"
//#include <iostream>
//#include "device_launch_parameters.h"
//#include "device_functions.h"
//#include <stdio.h>
//#include "constsmacros.h"
//#include <stdlib.h>
//#include <math.h>
//#include "ImageLoading.cu"
////#include "CUDAArray.cuh"
//#include <float.h>
//#include "OrientationField.cu"
//#include "Convolution.cu"
//#include "time.h"
//
//#define M_PI 3.14159265358979323846
//
//Point NewPoint(int x, int y)
//{
//	Point newP;
//	newP.x = x;
//	newP.y = y;
//	return newP;
//}
//
//int countOfEndings;
//
//void AddMinutiae(Minutiae* minutiaes, Minutiae minutiae, int* indexOfMinutiae)
//{
//	minutiaes[*indexOfMinutiae] = minutiae;
//	(*indexOfMinutiae)++;
//	//printf("Added new minutia. Type = %d\n", minutiae.type);
//	if (minutiae.type == 1) countOfEndings++;
//}
//
//bool OutOfImage(int x, int y, float* image, int* width, int* height)
//{
//	//160 - 192; 320 - 352
//	return (x < 0) || (y < 0) || (x >= *width) || (y >= *height);
//}
//
//int NewSection(int x, int y, Direction direction, float* image, float* orientField,
//	Point* section, float* sectionAngle, int* centerSection, bool* flag,
//	int sizeOfSection, int* width, int* height)
//{
//	int countOfPixels = 0;
//
//	//printf("=+=+=+=+=+=+=+=+=+=\n");
//	//printf("Making new section from point (%d, %d)\n", x, y);
//
//	for (int i = 0; i < sizeOfSection; i++)
//	{
//		section[i].x = -1;
//		section[i].y = -1;
//	}
//
//	int wing = sizeOfSection / 2;
//
//	int lEnd = wing, rEnd = lEnd;
//	bool rightE = false, leftE = false;
//
//	float angle = -orientField[y * (*width) + x];
//	//printf("Angle in this pixel: %f\n", angle);
//	angle += (float)M_PI_2;
//
//	section[wing] = NewPoint(x, y);
//	countOfPixels++;
//
//	for (int i = 1; i <= wing; i++)
//	{
//		int xs = (int)(x - i * cos(angle));
//		int ys = (int)(y + i * sin(angle) + 0.95);
//		int xe = (int)(x + i * cos(angle) + 0.95);
//		int ye = (int)(y - i * sin(angle));
//
//		//printf("Left pixel = (%d, %d); right pixel = (%d, %d)\n", xs, ys, xe, ye);
//
//		if (!OutOfImage(xs, ys, image, width, height) && (image[ys * (*width) + xs] < 20) && !rightE)
//		{
//			section[wing - i] = NewPoint(xs, ys);
//			rEnd--;
//			countOfPixels++;
//		}
//		else
//		{
//			rightE = true;
//		}
//
//		if (!OutOfImage(xe, ye, image, width, height) && (image[ye * (*width) + xe] < 20) && !leftE)
//		{
//			section[wing + i] = NewPoint(xe, ye);
//			lEnd++;
//			countOfPixels++;
//		}
//		else
//		{
//			leftE = true;
//		}
//
//		*centerSection = (lEnd + rEnd) / 2;
//	}
//	
//	x = section[*centerSection].x;
//	y = section[*centerSection].y;
//
//	//printf("New center: (%d, %d)\n", x, y);
//
//	angle = -orientField[y * (*width) + x];
//	angle += (float) direction * M_PI;
//	if (angle < 0) angle += 2.0 * M_PI;
//
//	if (abs(*sectionAngle - angle) > 0.2 && abs(*sectionAngle - angle) < 6) angle += M_PI;
//	while (angle > 2.0 * M_PI) angle -= 2 * M_PI;
//
//	//printf("Old angle: %f\nNew angle: %f\n", *sectionAngle, angle);
//
//	*sectionAngle = angle;
//
//	//printf("=+=+=+=+=+=+=+=+=+=\n");
//
//	return countOfPixels;
//}
//
//bool MakeStep(int* x, int* y, float* image, float* sectionAngle, Point* section,
//	int* sectionCenter, int step, int* width, int* height)
//{
//	*x = section[*sectionCenter].x;
//	*y = section[*sectionCenter].y;
//
//	float dx = (float)*x + step * cos(*sectionAngle);
//	float dy = (float)*y - step * sin(*sectionAngle);
//
//	*x = (int)(dx >= 0 ? dx + 0.5 : dx - 0.5);
//	*y = (int)(dy >= 0 ? dy + 0.5 : dy - 0.5);
//
//	if (OutOfImage(*x, *y, image, width, height)) return true;
//
//	return false;
//}
//
//int steps;
//
//void Paint(float* image, bool* visited, Point* oldSection, Point* newSection,
//	int size, int* width, int* height)
//{
//	Point queue[400];
//	Point v1, v2;
//
//	int left = 0, right = 0;
//
//	int x1 = -1, x2 = -1, y1 = -1, y2 = -1, x_a, y_a;
//
//	for (int i = 0; i < size; i++)
//	{
//		if (oldSection[i].x == -1) continue;
//
//		if (x1 == -1)
//		{
//			x1 = oldSection[i].x;
//			y1 = oldSection[i].y;
//		}
//
//		x2 = oldSection[i].x;
//		y2 = oldSection[i].y;
//
//		visited[y2 * (*width) + x2] = true;
//		queue[right] = NewPoint(x2, y2);
//		right++;
//	}
//
//	v1 = NewPoint(x2 - x1, y2 - y1);
//	x_a = x1; 
//	y_a = y1;
//
//	x1 = -1;
//	y1 = -1;
//	x2 = -1;
//	y2 = -1;
//
//	for (int i = 0; i < size; i++)
//	{
//		if (newSection[i].x == -1) continue;
//
//		if (x1 == -1)
//		{
//			x1 = newSection[i].x;
//			y1 = newSection[i].y;
//		}
//
//		x2 = newSection[i].x;
//		y2 = newSection[i].y;
//
//		visited[y2 * (*width) + x2] = true;
//	}
//
//	v2 = NewPoint(x2 - x1, y2 - y1);
//
//	if (v1.x*v2.x + v1.y*v2.y < 0)
//	{
//		x1 = x2;
//		y1 = y2;
//		v1.x = -v1.x;
//		v1.y = -v1.y;
//	}
//
//	while (abs(right - left) > 0)
//	{
//		for (int i = -2; i < 3; i++)
//			for (int j = -2; j < 3; j++)
//			{
//				if (i == 0 && j == 0) continue;
//
//				int x = queue[left].x + i;
//				int y = queue[left].y + j;
//
//				if (OutOfImage(x, y, image, width, height) || visited[y * (*width) + x] || image[y * (*width) + x] > 20) continue;
//
//				Point pointV1 = NewPoint(x_a - x, y_a - y);
//				Point pointV2 = NewPoint(x1 - x, y1 - y);
//
//				int skew1 = v1.x*pointV1.y - pointV1.x * v1.y >= 0 ? 1 : -1;
//				int skew2 = v2.x*pointV2.y - pointV2.x * v2.y >= 0 ? 1 : -1;
//
//				if (skew1*skew2 < 0)
//				{
//					queue[right] = NewPoint(x, y);
//					visited[y * (*width) + x] = true;
//					right++;
//					if (right == 400) right = 0;
//				}
//			} 
//
//		left++;
//		if (left == 400) left = 0;
//	}
//}
//
//void Paint2(float* image, bool* visited, Point* oldSection, float *sectionAngle,
//	int size, int* width, int* height)
//{
//	Point queue[40];
//
//	int rPointer = 0, lPointer = 0;
//
//	for (int i = 0; i < size; i++)
//	{
//		if (oldSection[i].x == -1) continue;
//
//		visited[oldSection[i].y * (*width) + oldSection[i].x] = true;
//		queue[rPointer] = oldSection[i];
//		rPointer++;
//	}
//
//	while (abs(rPointer - lPointer) > 0)
//	{
//		int x = queue[lPointer].x, y = queue[lPointer].y;
//
//		Point foo = NewPoint(x, y);
//		int bar = 0;
//
//		MakeStep(&x, &y, image, sectionAngle, &foo, &bar, 1, width, height);
//
//		for (int i = -1; i < 2; i++)
//			for (int j = -1; j < 2; j++)
//			{
//				if (OutOfImage(x + i, y + j, image, width, height)) continue;
//				if (image[(y + j) * (*width) + x + i] < 20 && !visited[(y + j) * (*width) + x + i])
//				{
//					visited[(y + j) * (*width) + x + i] = true;
//					queue[rPointer] = NewPoint(x + i, y + j);
//					rPointer++;
//					if (rPointer == 40) rPointer = 0;
//				}
//			}
//
//		lPointer++;
//		if (lPointer == 40) lPointer = 0;
//	}
//}
//
//MinutiaeType CheckStopCriteria(int x, int y, float* image, bool* visited, int* width, int threshold = 20)
//{
//	if (visited[y * (*width) + x])
//		return Intersection;
//
//	if (image[y * (*width) + x] > threshold)
//		return LineEnding;
//
//	return NotMinutiae;
//}
//
//void saveMyBmp_Paint(bool* visited, int index, int width, int height)
//{
//	int* img = (int*)malloc(width * height * sizeof(int));
//
//	for (int i = 0; i < height; i++)
//		for (int j = 0; j < width; j++)
//		{
//			if (visited[i * width + j]) img[i * width + j] = 255; else img[i * width + j] = 0;
//		}
//
//	char filename[80];
//	sprintf(filename, "res%d-%d.bmp", index, steps);
//
//	steps++;
//
//	saveBmp(filename, img, width, height);
//}
//
//int indexOfMinutiae = 0;
//
//void saveMyBmp(bool* visited, int* index, int width, int height)
//{
//	int* img = (int*)malloc(width * height * sizeof(int));
//
//	for (int i = 0; i < height; i++)
//		for (int j = 0; j < width; j++)
//		{
//			if (visited[i * width + j]) img[i * width + j] = 255; else img[i * width + j] = 0;
//		}
//
//	char filename[80];
//	sprintf(filename, "res%d-%d.bmp", indexOfMinutiae, *index);
//
//	saveBmp(filename, img, width, height);
//}
//
//
//void FollowLine(int x, int y, Direction direction, float* image, float* orientField,
//	bool* visited, Minutiae* minutiaes, Point* section, float* sectionAngle,
//	int* centerSection, bool* flag, int* sizeOfSection, int* step, int* width, 
//	int* height, int* indexOfMinutiae)
//{
//	int pixelsInSection = NewSection(x, y, direction, image, orientField, section,
//		sectionAngle, centerSection, flag, *sizeOfSection, width, height);
//	if (pixelsInSection == 1) return;
//
//	MinutiaeType type; steps = 0;
//	//int x, y;
//
//	Point* oldSection = new Point[*sizeOfSection];
//	bool outOfImage = false;
//
//	do
//	{
//		//printf("First/last section: \n");
//		for (int i = 0; i < *sizeOfSection; i++)
//		{
//			oldSection[i] = section[i];
//			//printf("(%d, %d) ", oldSection[i].x, oldSection[i].y);
//		}
//
//		//printf("\n");
//
//		outOfImage = MakeStep(&x, &y, image, sectionAngle, section, centerSection,
//			*step, width, height);
//
//		if (outOfImage)
//		{
//			Paint2(image, visited, oldSection, sectionAngle, *sizeOfSection, width, height);
//			return;
//		}
//
//		//printf("Step complited. New point: (%d, %d)\n", x, y);
//
//		type = CheckStopCriteria(x, y, image, visited, width);
//
//		NewSection(x, y, direction, image, orientField, section,
//			sectionAngle, centerSection, flag, *sizeOfSection, width, height);
//		if (section[*centerSection].x == -1)
//		{
//			Paint2(image, visited, oldSection, sectionAngle, *sizeOfSection, width, height);
//			return;
//		}
//
//		Paint(image, visited, oldSection, section, *sizeOfSection, 
//			width, height);
//		//saveMyBmp_Paint(visited, *indexOfMinutiae, *width, *height);
//	} while (type == NotMinutiae);
//
//	Minutiae newMinutiae;
//	newMinutiae.x = x;
//	newMinutiae.y = y;
//	newMinutiae.angle = *sectionAngle;
//	newMinutiae.type = type;
//
//	
//
//	AddMinutiae(minutiaes, newMinutiae, indexOfMinutiae);
//	
//	/*if (type == LineEnding) *///saveMyBmp(visited, indexOfMinutiae, *width, *height);
//}
//
//void FindMinutiae(float* image, float* orientField, bool* visited,
//	Minutiae* minutiaes, int sizeOfSection, int* width, 
//	int* height, int step, int colorThreshold = 20)
//{
//	Point* section = new Point[sizeOfSection];
//	float sectionAngle;
//	int centerSection;
//	bool flag = false;
//
//	int test = 0;
//
//	//160 - 192; 320 - 352
//	for (int i = 0; i < *width; i++)
//		for (int j = 0; j < *height; j++)
//		{
//			//On parallel version need add check out of image
//			if ((image[j * (*width) + i] >= colorThreshold) || (visited[j * (*width) + i]))
//				continue;
//
//			/*printf("================================================================\n");
//			printf("Starting point: (%d, %d)  //color = %f\n", i, j, image[j * (*width) + i]);
//			printf("     ***** Minutiae #%d *****     \n", indexOfMinutiae);
//			printf("Forward....\n");*/
//
//			sectionAngle = -orientField[j * (*width) + i];
//			if (sectionAngle < 0) sectionAngle += 2.0 * M_PI;
//			FollowLine(i, j, Forward, image, orientField, visited, minutiaes,
//				section, &sectionAngle, &centerSection, &flag, &sizeOfSection,
//				&step, width, height, &indexOfMinutiae);
//
//			flag = false;
//
//			/*printf("-----------------------------\n");
//			printf("     ***** Minutiae #%d *****     \n", indexOfMinutiae);
//			printf("Back....\n");*/
//			sectionAngle = -orientField[j * (*width) + i] + M_PI;
//			//if (sectionAngle > 2.0 * M_PI) sectionAngle -= M_PI;
//			FollowLine(i, j, Back, image, orientField, visited, minutiaes,
//				section, &sectionAngle, &centerSection, &flag, &sizeOfSection,
//				&step, width, height, &indexOfMinutiae);
//		}
//
//	//printf("Finded minutiaes: %d\n", indexOfMinutiae);
//}
//
//bool Start(Minutiae* minutiaeOut, float* source, int step, int lengthWings, int width, int height)
//{
//	/*DEBUG*/
//	//freopen("OUTPUT.log", "w", stdout);
//
//	int time = clock();
//
//	countOfEndings = 0;
//
//	bool* visited = (bool*)calloc(width * height, sizeof(bool));
//	int countOfMinutiae = 0;
//
//	float* orientFieldin = OrientationFieldInPixels(source, width, height);
//
//	/*printf("Angles:\n");
//	for (int i = 0; i < width; i++) {
//		for (int j = 0; j < height; j++)
//		{
//			if (source[j * width + i] < 15)
//				printf("%f ", orientFieldin[j * (width)+i]);
//			else
//				printf("0.000000 ", orientFieldin[j * (width)+i]);
//		}
//		printf("\n");
//	}*/
//
//	FindMinutiae(source, orientFieldin, visited, minutiaeOut, lengthWings * 2 + 1, &width, &height, step);
//
//	//minutiaeOut = minutiaes;
//
//	//printf("Endings: %d", countOfEndings);
//
//	DeleteDuplicate(minutiaeOut);
//
//	printf("Time: %d", clock() - time);
//
//	return false;
//}
//
////int main(int argc, char *argv[])
////{
////	/*DEBUG*/
////	freopen("OUTPUT.log", "w", stdout);
////
////
////	int width;
////	int height;
////	/*if (argc != 2)
////	{
////	printf("Need path to file");
////	return 0;
////	}*/
////	char* filename = "H:\\GitHub\\CUDA-Fingerprinting\\Code\\CUDAFingerprinting.GPU.RidgeLine\\res.bmp";  //Write your way to bmp file
////	int* img = loadBmp(filename, &width, &height);
////	float* source = (float*)malloc(height*width*sizeof(float));
////	for (int i = 0; i < height; i++)
////		for (int j = 0; j < width; j++)
////		{
////			source[(height - i - 1) * width + j] = (float)img[i * width + j];
////		}
////
////	//Minutiae* foo = (Minutiae*)malloc(sizeof(Minutiae) * width * height);
////
////	float* orientField = OrientationFieldInPixels(source, width, height);
////	Minutiae* minutiaes = (Minutiae*)calloc(width * height, sizeof(Minutiae));
////
////	/*for (int i = 0; i < width; i++)
////		printf("%f\n", orientField[30 * (width) + i]);*/
////
////	Start(minutiaes, source, 2, 3, width, height);
////
////	/*for (int i = 0; i < height; i++)
////	for (int j = 0; j < width; j++)
////	{
////	img[i * width + j] = res[i * width + j] ? 255 : 0;
////	}
////*/
////
////
////	//saveBmp("..\\rez.bmp", img, width, height);
////
////	return 0;
////}